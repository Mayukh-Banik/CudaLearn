#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }

    std::cout << "Number of CUDA devices: " << deviceCount << std::endl;

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device " << i << ": " << prop.name << std::endl;
        std::cout << "  Multiprocessors: " << prop.multiProcessorCount << std::endl;
        std::cout << "  Compute Capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "  Total Memory: " << (prop.totalGlobalMem / (1024 * 1024)) << " MB" << std::endl;
    }

    return 0;
}
