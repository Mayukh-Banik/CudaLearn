#include "hip/hip_runtime.h"
#include "defs/CreationRoutines.h"
#include <stdexcept>
#include <hip/hip_runtime_api.h>
#include <cmath>
#include <sstream>
#include "defs/CustomCudaMacros.h"

__global__ void deviceFillAllValuesWithConstant(double *Data, uint64_t NumElem, double val);

DoubleTensor *empty(uint64_t val, std::string Device)
{
    std::tuple<uint64_t, uint64_t> values = {val, 1};
    return empty(values, Device);
}

DoubleTensor *empty(std::vector<uint64_t> shape, std::string Device)
{
    if (shape.empty())
    {
        throw std::invalid_argument("How did you get here, list is empty.");
    }
    switch (shape.size())
    {
    case 0:
        throw std::invalid_argument("How did you get here, list is empty.");
        break;
    case 1:
        return empty(shape[0], Device);
    case 2:
        return empty(std::make_tuple(shape[0], shape[1]), Device);
    default:
        throw std::invalid_argument("List is too long.");
    }
    return nullptr;
}

DoubleTensor *empty(std::tuple<uint64_t, uint64_t> shape, std::string Device)
{
    std::vector<uint64_t> val = {std::get<0>(shape), std::get<1>(shape)};
    return new DoubleTensor(val, Device);
}

__global__ void eyeHelper(double *Data, const uint64_t (&input)[2], int64_t K)
{
    uint64_t index = threadIdx.x + blockDim.x * blockIdx.x;
    K = K + index;
    if (K >= 0 && K < input[1] && index < input[0])
    {
        Data[index * input[0] + K] = 1.0;
    }
}

DoubleTensor *eye(uint64_t N, uint64_t M, int64_t K, std::string Device)
{
    M = M == 0 ? N : M;
    std::vector<uint64_t> val(N, M);
    DoubleTensor *Tensor = new DoubleTensor(val, Device);
    if (Tensor->OnGPU)
    {
        ALLOCATE_BLOCKS_THREADS_SINGLE_TENSOR_ALL_VALUES(Tensor)
        eyeHelper<<<Blocks, Threads>>>(Tensor->Data, {NRows, NColumns}, K);
    }
    else
    {
        for (uint64_t i = 0; i < Tensor->Shape[0]; i++, K++)
        {
            if (K < 0)
            {
                continue;
            }
            else if (K >= Tensor->Shape[1])
            {
                break;
            }
            Tensor->Data[(i * Tensor->Shape[1]) + K] = 1.0;
        }
    }
    return Tensor;
}

DoubleTensor *identity(uint64_t N, std::string Device)
{
    return eye(N, 0, 0, Device);
}

DoubleTensor *zeros(uint64_t val, std::string Device)
{
    std::tuple<uint64_t, uint64_t> values = {val, 1};
    return zeros(values, Device);
}

DoubleTensor *zeros(std::vector<uint64_t> shape, std::string Device)
{
    if (shape.empty())
    {
        throw std::invalid_argument("How did you get here, list is empty.");
    }
    switch (shape.size())
    {
    case 0:
        throw std::invalid_argument("How did you get here, list is empty.");
        break;
    case 1:
        return zeros(shape[0], Device);
    case 2:
        return zeros(std::make_tuple(shape[0], shape[1]), Device);
    default:
        throw std::invalid_argument("List is too long.");
    }
    return nullptr;
}

DoubleTensor *zeros(std::tuple<uint64_t, uint64_t> shape, std::string Device)
{
    std::vector<uint64_t> val = {std::get<0>(shape), std::get<1>(shape)};
    DoubleTensor *Tensor = new DoubleTensor(val, Device);
    if (Tensor->OnGPU)
    {
        hipError_t err =
            CUDA_THROW_RUNTIME_ERROR_CHECK(err, hipMemset(Tensor->Data, 0, Tensor->ElementCount * Tensor->ItemSize);)
    }
    else
    {
        memset(Tensor->Data, 0, Tensor->ElementCount * Tensor->ItemSize);
    }
    return Tensor;
}

DoubleTensor *ones(uint64_t val, std::string Device)
{
    std::tuple<uint64_t, uint64_t> values = {val, 1};
    return ones(values, Device);
}

DoubleTensor *ones(std::vector<uint64_t> shape, std::string Device)
{
    if (shape.empty())
    {
        throw std::invalid_argument("How did you get here, list is empty.");
    }
    switch (shape.size())
    {
    case 0:
        throw std::invalid_argument("How did you get here, list is empty.");
        break;
    case 1:
        return ones(shape[0], Device);
    case 2:
        return ones(std::make_tuple(shape[0], shape[1]), Device);
    default:
        throw std::invalid_argument("List is too long.");
    }
    return nullptr;
}

DoubleTensor *ones(std::tuple<uint64_t, uint64_t> shape, std::string Device)
{
    std::vector<uint64_t> val = {std::get<0>(shape), std::get<1>(shape)};
    DoubleTensor *Tensor = new DoubleTensor(val, Device);
    if (Tensor->OnGPU)
    {
        ALLOCATE_BLOCKS_THREADS_SINGLE_TENSOR_ALL_VALUES(Tensor);
        deviceFillAllValuesWithConstant<<<Blocks, Threads>>>(Tensor->Data, Tensor->ElementCount, 1.0);
    }
    else
    {
        for (uint64_t i = 0; i < Tensor->ElementCount; i++)
        {
            Tensor->Data[i] = 1.0;
        }
    }
    return Tensor;
}

__global__ void deviceFillAllValuesWithConstant(double *Data, uint64_t NumElem, double val)
{
    uint64_t index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index < NumElem)
    {
        Data[index] = val;
    }
}

DoubleTensor *fill(uint64_t val, double vals, std::string Device)
{
    std::tuple<uint64_t, uint64_t> values = {val, 1};
    return fill(values, vals, Device);
}

DoubleTensor *fill(std::vector<uint64_t> shape, double vals, std::string Device)
{
    if (shape.empty())
    {
        throw std::invalid_argument("How did you get here, list is empty.");
    }
    switch (shape.size())
    {
    case 0:
        throw std::invalid_argument("How did you get here, list is empty.");
        break;
    case 1:
        return fill(shape[0], vals, Device);
    case 2:
        return fill(std::make_tuple(shape[0], shape[1]), vals, Device);
    default:
        throw std::invalid_argument("List is too long.");
    }
    return nullptr;
}

DoubleTensor *fill(std::tuple<uint64_t, uint64_t> shape, const double val, std::string Device)
{
    std::vector<uint64_t> vals = {std::get<0>(shape), std::get<1>(shape)};
    DoubleTensor *Tensor = new DoubleTensor(vals, Device);
    if (Tensor->OnGPU)
    {
        ALLOCATE_BLOCKS_THREADS_SINGLE_TENSOR_ALL_VALUES(Tensor)
        deviceFillAllValuesWithConstant<<<Blocks, Threads>>>(Tensor->Data, Tensor->ElementCount, val);
    }
    else
    {
        for (uint64_t i = 0; i < Tensor->ElementCount; i++)
        {
            Tensor->Data[i] = val;
        }
    }
    return Tensor;
}

DoubleTensor *array(double val, std::string Device)
{
    return new DoubleTensor(val, Device);
}

DoubleTensor *array(std::vector<std::vector<double>> values, std::string Device)
{
    DoubleTensor *Tensor = nullptr;

    Tensor = new DoubleTensor(values, Device);

    return Tensor;
}

DoubleTensor *array(nanobind::ndarray<double, nanobind::shape<-1, -1>, nanobind::any_contig> array, bool copy)
{
    std::ostringstream S;
    if (array.DeviceType == nanobind::device::cpu::value)
    {
        S << "cpu";
    }
    else
    {
        S << "cuda:" << array.device_id();
    }
    DoubleTensor *tensor = new DoubleTensor(S.str());
    tensor->ElementCount = array.size();
    tensor->Shape[0] = array.shape_ptr()[0];
    tensor->Shape[1] = array.shape_ptr()[1];
    tensor->Strides[0] = array.stride_ptr()[0];
    tensor->Strides[1] = array.stride_ptr()[1];
    tensor->Order[0] = array.Order;
    if (copy)
    {
        if (tensor->OnGPU)
        {
            hipError_t err;
            CUDA_THROW_RUNTIME_ERROR_CHECK(err, hipMemcpy(tensor->Data, array.data(), tensor->ElementCount * tensor->ItemSize, hipMemcpyDeviceToDevice))
        }
        else
        {
            memcpy(tensor->Data, array.data(), tensor->ElementCount * tensor->ItemSize);
        }
    }
    else
    {
        tensor->Data = array.data();
    }
    return tensor;
}

DoubleTensor *array(nanobind::ndarray<double, nanobind::shape<-1>, nanobind::any_contig> array, bool copy)
{
    std::ostringstream S;
    if (array.DeviceType == nanobind::device::cpu::value)
    {
        S << "cpu";
    }
    else
    {
        S << "cuda:" << array.device_id();
    }
    DoubleTensor *tensor = new DoubleTensor(S.str());
    tensor->ElementCount = array.size();
    tensor->Shape[0] = array.shape_ptr()[0];
    tensor->Shape[1] = 1;
    tensor->Strides[0] = array.stride_ptr()[0];
    tensor->Strides[1] = 1;
    tensor->Order[0] = array.Order;
    if (copy)
    {
        if (tensor->OnGPU)
        {
            hipError_t err = hipMemcpy(tensor->Data, array.data(), tensor->ElementCount * tensor->ItemSize, hipMemcpyDeviceToDevice);
            if (err != hipSuccess)
            {
                throw std::runtime_error("Error copying over data in GPU memory, try setting copy to false.");
            }
        }
        else
        {
            memcpy(tensor->Data, array.data(), tensor->ElementCount * tensor->ItemSize);
        }
    }
    else
    {
        tensor->Data = array.data();
    }
    return tensor;
}