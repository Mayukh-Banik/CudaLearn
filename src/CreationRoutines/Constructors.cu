#include "defs/DoubleTensor.h"

#include <hip/hip_runtime_api.h>
#include <cstdlib>
#include <stdexcept>
#include <cstring>
#include <hip/hip_runtime.h>

/**
 * @brief Tensor from a scalar value, default CPU
 *
 * @details Creates a Tensor from the supplied double value,
 *  internally its always stored as a Matrix. Strides are special to be 0,0
 *  but once exported to a numpy array it will be (8,8)
 */
DoubleTensor::DoubleTensor(double val, std::string Device)
{
    this->Shape = {1, 1};
    this->Strides = {0, 0};
    this->ElementCount = 1;
    this->OnGPU = Device != "cpu";
    this->Device = Device;
    if (OnGPU)
    {
        hipError_t err = hipMalloc((void **)&Data, ItemSize);
        if (err != hipSuccess)
        {
            throw std::runtime_error(hipGetErrorString(err));
        }
        err = hipMemcpy(this->Data, (void *)&val, this->ItemSize, hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            throw std::runtime_error(hipGetErrorString(err));
        }
    }
    else
    {
        this->Data = (double *)malloc(this->ItemSize);
        if (this->Data == NULL)
        {
            throw std::runtime_error("Memory allocation failed: " + errno);
        }
        this->Data[0] = val;
    }
}

DoubleTensor::DoubleTensor(std::vector<uint64_t> Shape, std::string Device)
{
    switch (Shape.size())
    {
    case 1:
        Shape.push_back(1);
        break;
    case 2:
        break;
    default:
        throw std::invalid_argument("Number of dims > 2.");
    }
    if (Shape[0] == 0 || Shape[1] == 0)
    {
        throw std::invalid_argument("Shape vals must be >0.");
    }
    this->Shape = Shape;
    this->Strides = {this->Shape[1] * this->ItemSize, this->ItemSize};
    this->ElementCount = this->Shape[0] * this->Shape[1];
    this->OnGPU = Device != "cpu";
    this->Device = Device;
    if (OnGPU)
    {
        hipError_t err = hipMalloc((void **)&Data, this->ItemSize * this->ElementCount);
        if (err != hipSuccess)
        {
            throw std::runtime_error(hipGetErrorString(err));
        }
    }
    else
    {
        this->Data = (double *)malloc(this->ItemSize * this->ElementCount);
        if (this->Data == NULL)
        {
            throw std::runtime_error("Memory allocation failed: " + errno);
        }
    }
}

DoubleTensor::DoubleTensor(std::vector<std::vector<double>> Value, std::string Device)
{
    if (Value.empty())
    {
        throw std::invalid_argument("The vector is empty.");
    }

    size_t cols = Value[0].size();
    for (const std::vector<double> row : Value)
    {
        if (row.size() != cols)
        {
            throw std::invalid_argument("Inconsistent matrix dimensions: All rows must have the same number of columns.");
        }
    }

    this->Shape = {Value.size(), Value[0].size()};
    this->Strides = {this->Shape[1] * this->ItemSize, this->ItemSize};
    this->ElementCount = this->Shape[0] * this->Shape[1];
    this->OnGPU = Device != "cpu";
    this->Device = Device;
    if (OnGPU)
    {
        hipError_t err = hipMalloc((void **)&Data, this->ItemSize * this->ElementCount);
        if (err != hipSuccess)
        {
            throw std::runtime_error(hipGetErrorString(err));
        }
        for (size_t i = 0, index = 0; i < Value.size(); i++, index = index + Value[i].size())
        {
            err = hipMemcpy(Data + index, Value[i].data(), Value[i].size() * this->ItemSize, hipMemcpyHostToDevice);
            if (err != hipSuccess)
            {
                throw std::runtime_error(hipGetErrorString(err));
            }
        }
    }
    else
    {
        this->Data = (double *)malloc(this->ItemSize * this->ElementCount);
        if (this->Data == NULL)
        {
            throw std::runtime_error("Memory allocation failed: " + errno);
        }
        for (size_t i = 0, index = 0; i < Value.size(); i++, index = index + Value[i].size())
        {
            std::memcpy(this->Data + index, Value[i].data(), Value[i].size() * this->ItemSize);
        }
    }
}

DoubleTensor::~DoubleTensor()
{
    if (OnGPU)
    {
        hipFree(this->Data);
    }
    else
    {
        free(this->Data);
    }
}

inline void DoubleTensor::getDeviceProperties()
{
    hipDeviceProp_t prop;
    int device;
    if (hipGetDevice(&device) == hipSuccess)
    {
        hipGetDeviceProperties(&prop, device);

        this->deviceProperties.MaxGridSize[0] = prop.maxGridSize[0];
        this->deviceProperties.MaxGridSize[1] = prop.maxGridSize[1];
        this->deviceProperties.MaxGridSize[2] = prop.maxGridSize[2];

        this->deviceProperties.MaxThreadsDim[0] = prop.maxThreadsDim[0];
        this->deviceProperties.MaxThreadsDim[1] = prop.maxThreadsDim[1];
        this->deviceProperties.MaxThreadsDim[2] = prop.maxThreadsDim[2];

        this->deviceProperties.MaxThreadsPerBlock = prop.maxThreadsPerBlock;
        this->deviceProperties.WarpSize = prop.warpSize;
    }
}
